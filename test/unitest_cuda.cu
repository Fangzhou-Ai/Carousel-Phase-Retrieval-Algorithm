#include <gtest/gtest.h>
#include "../include/CPRA.hpp"


TEST(CUDATEST, Test_IO_Host_BINARY)
{
    CPRA::Cpra<float, CPRA::IMPL_TYPE::CUDA> obj(1, 1, 1, 1);
    float* output_ptr = (float*)obj.allocate(sizeof(float) * 1000);
    for(int i = 0; i < 1000; i++)
        output_ptr[i] = i;
    EXPECT_EQ(obj.WriteMatrixToFile("/tmp/test_output.bin", output_ptr, 10, 10, 10), true);

    float* Input_ptr = (float*)obj.allocate(sizeof(float) * 1000);
    EXPECT_EQ(obj.ReadMatrixFromFile("/tmp/test_output.bin", Input_ptr, 10, 10, 10), true);
    for(int i = 0; i < 1000; i++)
        EXPECT_EQ(Input_ptr[i], i);

    obj.deallocate(output_ptr);
    obj.deallocate(Input_ptr);
}

TEST(CUDATEST, Test_Merge_Norm)
{
    CPRA::Cpra<float, CPRA::IMPL_TYPE::CUDA> obj(1, 1, 1, 1);
    std::complex<float>* input = (std::complex<float>*)obj.allocate(sizeof(std::complex<float>) * 10);
    for(auto i = 0; i < 10; i++)
    {
        input[i].real(10);
        input[i].imag(10);
    }
    obj.impl_->MergeAddData(input, input, 1, 1, 10);

    hipDeviceSynchronize(); // We need to sync here to make sure kernel is finished
    for(auto i = 0; i < 10; i++)
    {
        EXPECT_EQ(input[i].real(), 20); 
        EXPECT_EQ(input[i].imag(), 20);   
    }
    
    obj.impl_->Normalization(input, 20, 10);
    hipDeviceSynchronize(); // We need to sync here to make sure kernel is finished
    for(auto i = 0; i < 10; i++)
    {
        EXPECT_EQ(input[i].real(), 1); 
        EXPECT_EQ(input[i].imag(), 1);   
    }


    obj.deallocate(input);
}